#include "hip/hip_runtime.h"
// This kernel code based on CUDPP.  Please see the notice in
// LICENSE_CUDPP.txt.

typedef unsigned int uint;

typedef struct __builtin_align__(16) {
    uint4 a;
    uint4 b;
  } my_uint8;

#if SORT_VECTOR == 2
#define SORTVECTYPE uint2
#elif SORT_VECTOR == 4
#define SORTVECTYPE uint4
#elif SORT_VECTOR == 8
#define SORTVECTYPE my_uint8
#endif

#if SCAN_VECTOR == 2
#define SCANVECTYPE uint2
#elif SCAN_VECTOR == 4
#define SCANVECTYPE uint4
#elif SCAN_VECTOR == 8
#define SCANVECTYPE my_uint8
#endif

//----------------------------------------------------------------------------
//
// radixSortBlocks sorts all blocks of data independently in shared
// memory.  Each thread block (CTA) sorts one block of SORT_VECTOR*CTA_SIZE elements
//
// The radix sort is done in two stages.  This stage calls radixSortBlock
// on each block independently, sorting on the basis of bits
// (startbit) -> (startbit + nbits)
//----------------------------------------------------------------------------

extern "C" __global__ void radixSortBlocks(SORTVECTYPE* keysOut, SORTVECTYPE* valuesOut,
                              SORTVECTYPE* keysIn,  SORTVECTYPE* valuesIn)
{
    // Get Indexing information
    const uint i = threadIdx.x + (blockIdx.x * blockDim.x);
    const uint tid = threadIdx.x;
    const uint localSize = blockDim.x;

    // Load keys and vals from global memory
    SORTVECTYPE key, value;
    key = keysIn[i];
    value = valuesIn[i];
    
    if (i == 0) printf("beginning\n");
    if (i < 10) {
      printf("%d  %d  %d\n", i, key.x, value.x);
    }


    // For each of the 4 bits
    {

        // Read keys out of local mem into registers, in prep for
        // write out to global mem
#if SORT_VECTOR == 2
        key.x = key.x + 2;
        key.y = key.y + 2;
#elif SORT_VECTOR == 4
        key.x = key.x + 2;
        key.y = key.y + 2;
        key.z = key.z + 2;
        key.w = key.w + 2;
#elif SORT_VECTOR == 8
        key.a.x = key.a.x + 2;
        key.a.y = key.a.y + 2;
        key.a.z = key.a.z + 2;
        key.a.w = key.a.w + 2;
        key.b.x = key.b.x + 2;
        key.b.y = key.b.y + 2;
        key.b.z = key.b.z + 2;
        key.b.w = key.b.w + 2;
#endif
        __syncthreads();

    keysOut[i]   = key;
    valuesOut[i] = value;
    if (i == 0) printf("end\n");
    if (i < 10) {
      printf("%d  %d  %d\n", i, key.x, value.x);
    }
    }
}
